#include "hip/hip_runtime.h"
#include "cudafilter.hpp"
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#define THREADS_PER_DIM 32
#define CUDA_ERR_HANDLER(err) cudaErrorHandler(err, __FILE__, __LINE__)

__constant__ float cfilter[FILTER_SIZE * FILTER_SIZE];

static void cudaErrorHandler(hipError_t err, const char *file, int line) {
   if(err != hipSuccess) {
      std::cerr<<hipGetErrorString(err)<<" on line "<<line<<" : "<<file<<std::endl;
      exit(EXIT_FAILURE);
   }
}

__device__ void convolutionFilter(Image image, Image result, int x, int y) {
   
   /*
   
   float3 bgr;

   //multiply every value of the filter with corresponding image pixel 
   for(int filterX = 0; filterX < FILTER_SIZE; filterX++) {
      for(int filterY = 0; filterY < FILTER_SIZE; filterY++) {
         int imageX = x - FILTER_SIZE / 2 + filterX;
         int imageY = y - FILTER_SIZE / 2 + filterY;
         if(imageX < 0 || imageX >= image.width ||
            imageY < 0 || imageY >= image.height)
            continue;
         
         float filterVal = cfilter[filterY * FILTER_SIZE + filterX];
         bgr.x += image.at(imageY, imageX, BLUE) * filterVal;
         bgr.y += image.at(imageY, imageX, GREEN) * filterVal;
         bgr.z += image.at(imageY, imageX, RED) * filterVal;
      } 
   }

   */
   
   
   
   float3 bgr;

   __shared__ unsigned sharedImage[THREADS_PER_DIM][THREADS_PER_DIM * 3];
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   sharedImage[ty][3*tx] = image.at(y, x, BLUE);
   sharedImage[ty][3*tx+1] = image.at(y, x, GREEN);
   sharedImage[ty][3*tx+2] = image.at(y, x, RED);

   __syncthreads();

  // multiply every value of the filter with corresponding image pixel 
   for(int filterX = 0; filterX < FILTER_SIZE; filterX++) {
      for(int filterY = 0; filterY < FILTER_SIZE; filterY++) {

         unsigned shX = tx - FILTER_SIZE / 2 + filterX;
         unsigned shY = ty - FILTER_SIZE / 2 + filterY;
         float filterVal = cfilter[filterY * FILTER_SIZE + filterX];
         
         shX >= THREADS_PER_DIM && (shX = THREADS_PER_DIM - 1);
         shY >= THREADS_PER_DIM && (shY = THREADS_PER_DIM - 1);
         shX < 0 && (shX = 0);
         shY < 0 && (shY = 0);
         

         bgr.x += sharedImage[shY][3 * shX] * filterVal; 
         bgr.y += sharedImage[shY][3 * shX + 1] * filterVal;
         bgr.z += sharedImage[shY][3 * shX + 2] * filterVal; 
      } 
   }
   
   //truncate values smaller than zero and larger than 255 
   result.at(y, x, BLUE) = min(max(int(1 * bgr.x + 0), 0), 255); 
   result.at(y, x, GREEN) = min(max(int(1 * bgr.y + 0), 0), 255); 
   result.at(y, x, RED) = min(max(int(1 * bgr.z + 0), 0), 255); 
}

__global__ void filterKernal(Image image, Image result) {
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   int x = blockIdx.x * blockDim.x + threadIdx.x;

   if(y < image.height && x < image.width) {
      convolutionFilter(image, result, x, y);
   }
}

CudaFilter::CudaFilter(Image image, Filter filter)
   :image(image), devImage(image), devResult(image){
      hipMemcpyToSymbol(HIP_SYMBOL(cfilter), filter.data, FILTER_SIZE * FILTER_SIZE * sizeof(float));
      toDevice((void**)&devImage.data, image.data, image.size);
      toDevice((void**)&devResult.data, image.data, image.size);
   //   toDevice((void**)&devFilter.data, filter.data, filter.rows * filter.cols * sizeof(float));
}

CudaFilter::~CudaFilter() {
   assert(image.size == devResult.size);
   toHost(image.data, devResult.data, image.size);
   CUDA_ERR_HANDLER(hipFree(devImage.data));
   CUDA_ERR_HANDLER(hipFree(devResult.data));
   //CUDA_ERR_HANDLER(hipFree(devFilter.data));
}

void CudaFilter::toDevice(void **dev, void *host, int bytes) {
   CUDA_ERR_HANDLER(hipMalloc(dev, bytes));
   CUDA_ERR_HANDLER(hipMemcpy(*dev, host, bytes, hipMemcpyHostToDevice));
}

void CudaFilter::toHost(void *host, void *dev, int bytes) {
   CUDA_ERR_HANDLER(hipMemcpy(host, dev, bytes, hipMemcpyDeviceToHost));
}

void CudaFilter::applyFilter() {
   dim3 threadsPerBlock(THREADS_PER_DIM, THREADS_PER_DIM);
   dim3 blocksPerGrid((image.width + THREADS_PER_DIM - 1) / THREADS_PER_DIM, 
                      (image.height + THREADS_PER_DIM - 1) / THREADS_PER_DIM);

   filterKernal<<<blocksPerGrid, threadsPerBlock>>>(devImage, devResult);
   hipDeviceSynchronize();
   CUDA_ERR_HANDLER(hipGetLastError());
}

float CudaFilter::operator() () {
   float ms;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   applyFilter();

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&ms, start, stop);
   return ms;
}