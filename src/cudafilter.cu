#include "hip/hip_runtime.h"
#include "cudafilter.hpp"
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include <iostream>

#define THREADS_PER_DIM 32
#define CUDA_ERR_HANDLER(err) cudaErrorHandler(err, __FILE__, __LINE__)

static void cudaErrorHandler(hipError_t err, const char *file, int line) {
   if(err != hipSuccess) {
      std::cerr<<hipGetErrorString(err)<<" on line "<<line<<" : "<<file<<std::endl;
      exit(EXIT_FAILURE);
   }
}

__device__ void convolutionFilter(Image image, Image result, Filter filter, int x, int y) {
   float3 bgr;

   //multiply every value of the filter with corresponding image pixel 
   for(int filterX = 0; filterX < filter.cols; filterX++) {
      for(int filterY = 0; filterY < filter.rows; filterY++) {
         int imageX = x - filter.cols / 2 + filterX;
         int imageY = y - filter.rows / 2 + filterY;
         if(imageX < 0 || imageX >= image.height ||
            imageY < 0 || imageY >= image.width)
            continue;
         
         float filterVal = filter[filterX][filterY];
         bgr.x += image.at(imageX, imageY, BLUE) * filterVal;
         bgr.y += image.at(imageX, imageY, GREEN) * filterVal;
         bgr.z += image.at(imageX, imageY, RED) * filterVal;
      } 
   }
   
   /*
   float3 bgr;

   __shared__ unsigned sharedImage[THREADS_PER_DIM][THREADS_PER_DIM * 3];
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   sharedImage[tx][3*ty] = image.at(x, y, BLUE);
   sharedImage[tx][3*ty+1] = image.at(x, y, GREEN);
   sharedImage[tx][3*ty+2] = image.at(x, y, RED);

   __syncthreads();

   //multiply every value of the filter with corresponding image pixel 
   for(int filterX = 0; filterX < filter.cols; filterX++) {
      for(int filterY = 0; filterY < filter.rows; filterY++) {
         int imageX = tx - filter.cols / 2 + filterX;
         int imageY = ty - filter.rows / 2 + filterY;
         imageX = min(max(0, imageX), THREADS_PER_DIM - 1);
         imageY = min(max(0, imageY), THREADS_PER_DIM - 1);
         
         float filterVal = filter[filterX][filterY];
         bgr.x += sharedImage[imageX][3 * imageY] * filterVal;//image.at(imageX, imageY, BLUE) * filterVal;
         bgr.y += sharedImage[imageX][3 * imageY + 1] * filterVal; //image.at(imageX, imageY, GREEN) * filterVal;
         bgr.z += sharedImage[imageX][3 * imageY + 2] * filterVal; //image.at(imageX, imageY, RED) * filterVal;
      } 
   }
   */


   //truncate values smaller than zero and larger than 255 
   result.at(x, y, BLUE) = min(max(int(filter.factor * bgr.x + filter.bias), 0), 255); 
   result.at(x, y, GREEN) = min(max(int(filter.factor * bgr.y + filter.bias), 0), 255); 
   result.at(x, y, RED) = min(max(int(filter.factor * bgr.z + filter.bias), 0), 255); 
}

__global__ void filterKernal(Image image, Image result, Filter filter) {
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int col = blockIdx.x * blockDim.x + threadIdx.x;

   if(row < image.height && col < image.width) {
      convolutionFilter(image, result, filter, row, col);
   }
}

CudaFilter::CudaFilter(Image image, Filter filter)
   :image(image), filter(filter) {}

CudaFilter::~CudaFilter() {
   for(int i=0; i<devmem.size(); i++)
      hipFree(devmem.at(i));
}

void CudaFilter::toDevice(void **dev, void *host, int bytes) {
   CUDA_ERR_HANDLER(hipMalloc(dev, bytes));
   CUDA_ERR_HANDLER(hipMemcpy(*dev, host, bytes, hipMemcpyHostToDevice));
   devmem.push_back(*dev);
}

void CudaFilter::toHost(void *host, void *dev, int bytes) {
   CUDA_ERR_HANDLER(hipMemcpy(host, dev, bytes, hipMemcpyDeviceToHost));
}

void CudaFilter::applyFilter() {
   Image devImage(image), devResult(image);
   Filter devFilter(filter);

   toDevice((void**)&devImage.data, image.data, image.size);
   toDevice((void**)&devResult.data, image.data, image.size);
   toDevice((void**)&devFilter.data, filter.data, filter.rows * filter.cols * sizeof(float));

   dim3 threadsPerBlock(THREADS_PER_DIM, THREADS_PER_DIM);
   dim3 blocksPerGrid((image.width + THREADS_PER_DIM - 1) / THREADS_PER_DIM, 
                      (image.height + THREADS_PER_DIM - 1) / THREADS_PER_DIM);

   filterKernal<<<blocksPerGrid, threadsPerBlock>>>(devImage, devResult, devFilter);
   CUDA_ERR_HANDLER(hipGetLastError());

   assert(image.size == devResult.size);
   toHost(image.data, devResult.data, image.size);
}

float CudaFilter::operator() () {
   float ms;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   applyFilter();

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&ms, start, stop);
   return ms;
}